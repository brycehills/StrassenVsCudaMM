
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    //declare shared memory/thread ints
    __shared__ float ds_M[TILE_SIZE][TILE_SIZE];
    __shared__ float ds_N[TILE_SIZE][TILE_SIZE];


    //declare matrix index vars
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    ///define row/col
    int Row = by * blockDim.y + ty;
    int Col = bx * blockDim.x + tx;

    
    // load data from A,B into shared mem + boundary checking & zero padding
    for(int p = 0; p < (k-1)/TILE_SIZE-1;p++)
    {
    	if(Row < m && (p * TILE_SIZE + tx < k)) //load M - within boundary;  note: (a = m x k)
	{
	    ds_M[ty][tx] = A[Row * k + p * TILE_SIZE + tx];
	}
	else // pad 0
	{
	   ds_M[ty][tx] = 0.0;
	}
	if((p*TILE_SIZE + ty < k) && Col < n) // load N - within boundary; note: (b = k x n)
	{
	   ds_N[ty][tx] = B[(p*TILE_SIZE + ty) * k + Col];
	}
	else // pad 0
	{
	    ds_N[ty][tx] = 0.0;
	}
		
    }

    //perform computation (inner product) into p variable

    //sync 
    // loop to load p variable into C    
    /*************************************************************************/
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE

    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
	
    /*************************************************************************/
}


