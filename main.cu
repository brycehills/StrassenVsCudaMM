#include "hip/hip_runtime.h"
#include <stdio.h>
#include<time.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"
#include <omp.h>
#include <iostream>
#include "mm.cpp"

//using namespace std;

int main (int argc, char *argv[])
{
    Timer timer;
    hipError_t cuda_ret;

    //get size from user -- default is 8x8
    int input_size = 8;
    cout << "Enter Matrix Size (power of 2): ";
    cin >> input_size;
    cout << endl << "Matrix Size = " << input_size << endl;

    //*********************************************************************
    // ~~ final project driver code ~~
    //----------------------------------
    vector<float> im (input_size);
    // allocate vector matrices -- on heap
    vector<vector<float> > a(input_size,im);
    vector<vector<float> > b(input_size,im);
    vector<vector<float> > r(input_size,im);
    vector<vector<float> > r2(input_size,im);
    // populate matrices w random values
    for(int i = 0; i < input_size; i++){
	for(int j = 0; j< input_size; j++){
		a[i][j] = (rand()%100)/100.00;
		b[i][j] = (rand()%100)/100.00;
	}
     }

    //--------------------------------------------------------------------
    //naive mm driver code
    //--------------------------------------------------------------------
    cout << "starting naive mm" << endl;
    clock_t start = clock();
    mm(a,b,r,input_size);
    //PrintMatrix(r,input_size);
    clock_t end = clock();
    double timetaken = double(end - start)/CLOCKS_PER_SEC;
    printf("Time measured: %.3f seconds.\n", timetaken);
    cout << endl << endl;
    //--------------------------------------------------------------------
    // strassen driver code
    //--------------------------------------------------------------------
    cout << "starting stassen mm" << endl;
    // start strass timer
    clock_t start1 = clock();
    Strassen(a,b,r2,input_size);
    clock_t end1 = clock();
    double timetaken_strass = double(end1 - start1)/CLOCKS_PER_SEC;
    printf("Time measured for strassen mm: %.3f seconds.\n", timetaken_strass);
    //PrintMatrix(r2,input_size);
    cout << endl << endl;

    //*********************************************************************

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    float *A_d, *B_d, *C_d;
    size_t A_sz, B_sz, C_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        matArow = 1000;
        matAcol = matBrow = 1000;
        matBcol = 1000;
    } else if (argc == 2) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[1]);
        matBcol = atoi(argv[1]);
    } else if (argc == 4) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[2]);
        matBcol = atoi(argv[3]);
    } else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./sgemm-tiled                # All matrices are 1000 x 1000"
      "\n    Usage: ./sgemm-tiled <m>            # All matrices are m x m"
      "\n    Usage: ./sgemm-tiled <m> <k> <n>    # A: m x k, B: k x n, C: m x n"
      "\n");
        exit(0);
    }
   
    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;

    A_h = (float*) malloc( sizeof(float)*A_sz );
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    B_h = (float*) malloc( sizeof(float)*B_sz );
    for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    C_h = (float*) malloc( sizeof(float)*C_sz );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", matArow, matAcol,
        matBrow, matBcol, matArow, matBcol);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    /*************************************************************************/
    //INSERT CODE HERE
    hipMalloc((void**)&A_d,sizeof(float)*A_sz);
    hipMalloc((void**)&B_d,sizeof(float)*B_sz);
    hipMalloc((void**)&C_d,sizeof(float)*C_sz);
    /*************************************************************************/
	
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);
    time_t start2 = clock();	
    /*************************************************************************/
    //INSERT CODE HERE
    hipMemcpy(A_d,A_h,sizeof(float)*A_sz,hipMemcpyHostToDevice);
    hipMemcpy(B_d,B_h,sizeof(float)*B_sz,hipMemcpyHostToDevice);

    /*************************************************************************/
    
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel using standard sgemm interface ---------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
    basicSgemm(matArow, matBcol, matBrow, A_d, B_d, C_d);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    /*************************************************************************/
    //INSERT CODE HERE
    hipMemcpy(C_h,C_d,sizeof(float)*C_sz,hipMemcpyDeviceToHost);
    /*************************************************************************/
    time_t end2 = clock();
    double gpu_time = double(end2 - start2)/CLOCKS_PER_SEC;
    hipDeviceSynchronize();
    //stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("\nTime taken for CUDA MM:  %.3f seconds.\n", gpu_time);
    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, matArow, matAcol, matBcol); //commented out for timing results


    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    /*************************************************************************/
    //INSERT CODE HERE
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);   
    /*************************************************************************/

    return 0;
}

